//
// Created by wxm on 2023/7/26.
//

#include "GPUS_Function.cuh"
void enableP2P (int ngpus)
{
    for (int i=0; i<ngpus; i++)
    {
        CHECK(hipSetDevice(i));

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            CHECK(hipDeviceCanAccessPeer(&peer_access_available, i, j));

            if (peer_access_available) CHECK(hipDeviceEnablePeerAccess(j, 0));
            else
            {
                cout<<"can't acess "<<endl;
            }
        }
    }
}

bool isCapableP2P(int ngpus)
{
    hipDeviceProp_t prop[ngpus];
    int iCount = 0;

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipGetDeviceProperties(&prop[i], i));

        if (prop[i].major >= 2) iCount++;

        printf("> GPU%d: %s %s capable of Peer-to-Peer access\n", i,
               prop[i].name, (prop[i].major >= 2 ? "is" : "not"));
        fflush(stdout);
    }

    if(iCount != ngpus)
    {
        printf("> no enough device to run this application\n");
        fflush(stdout);
    }

    return (iCount == ngpus);
}


//void transfer_data(const int GPU_ID, const int GPUS, int **exchange_flag, int **halo_size, IHP_SIZE &ihpSize, real **data, hipStream_t **st_halo)
//{
//
//    for(int j=0; j<GPUS; j++)
//    {
//        if (exchange_flag[GPU_ID][j]==1)
//        {
//            CHECK(hipMemcpyAsync(dataGpuCell[j].data_padding[GPU_ID], dataGpuCell[j].data_halo[GPU_ID],
//                                  halo_size[][j]*sizeof(real), hipMemcpyDeviceToDevice, st_halo[j]));
//        }
//    }
//
//}

void device_sync(int GPUS)
{
    for (int i=0; i<GPUS; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipDeviceSynchronize());
    }
}


